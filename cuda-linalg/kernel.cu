#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "kernel.cuh"
#include <algorithm>

__global__ void addKernel(double* c, const double* a, const double* b, size_t size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	for (size_t j = 0; j < size; j += blockDim.x)
	{
		if (i + j < size)
			c[i + j] = a[i + j] + b[i + j];
	}
}

__global__ void subKernel(double* c, const double* a, const double* b, size_t size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	for (size_t j = 0; j < size; j += blockDim.x)
	{
		if (i + j < size)
			c[i + j] = a[i + j] - b[i + j];
	}
}

__global__ void mulKernel(double* c, const double* a, double* b, size_t size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	for (size_t j = 0; j < size; j += blockDim.x)
	{
		if (i + j < size)
			c[i + j] = a[i + j] * (*b);
	}
}

void CUDA::Vector::add(double* result, const double* left, const double* right, size_t size)
{
	double* dev_left = nullptr;
	double* dev_right = nullptr;
	double* dev_result = nullptr;

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

	hipMalloc((void**)&dev_left, size * sizeof(double));
	hipMalloc((void**)&dev_right, size * sizeof(double));
	hipMalloc((void**)&dev_result, size * sizeof(double));

	hipMemcpy(dev_left, left, size * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_right, right, size * sizeof(double), hipMemcpyHostToDevice);
	
	addKernel << < std::min(prop.maxGridSize[0], (int)size / prop.maxThreadsPerBlock + 1), std::min((int)size, prop.maxThreadsPerBlock) >> > (dev_result, dev_left, dev_right, size);
	
	hipMemcpy(result, dev_result, size * sizeof(double), hipMemcpyDeviceToHost);
	
	hipFree(dev_left);
	hipFree(dev_right);
	hipFree(dev_result);
}

void CUDA::Vector::sub(double* result, const double* left, const double* right, size_t size)
{
	double* dev_left = nullptr;
	double* dev_right = nullptr;
	double* dev_result = nullptr;

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

	hipMalloc((void**)&dev_left, size * sizeof(double));
	hipMalloc((void**)&dev_right, size * sizeof(double));
	hipMalloc((void**)&dev_result, size * sizeof(double));

	hipMemcpy(dev_left, left, size * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_right, right, size * sizeof(double), hipMemcpyHostToDevice);

	subKernel << < std::min(prop.maxGridSize[0], (int)size / prop.maxThreadsPerBlock + 1), std::min((int)size, prop.maxThreadsPerBlock) >> > (dev_result, dev_left, dev_right, size);

	hipMemcpy(result, dev_result, size * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(dev_left);
	hipFree(dev_right);
	hipFree(dev_result);
}

void CUDA::Vector::mul(double* result, const double* left, double right, size_t size)
{
	double* dev_left = nullptr;
	double* dev_right = nullptr;
	double* dev_result = nullptr;

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

	hipMalloc((void**)&dev_left, size * sizeof(double));
	hipMalloc((void**)&dev_right, sizeof(double));
	hipMalloc((void**)&dev_result, size * sizeof(double));

	hipMemcpy(dev_left, left, size * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_right, &right, sizeof(double), hipMemcpyHostToDevice);

	mulKernel << < std::min(prop.maxGridSize[0], (int)size / prop.maxThreadsPerBlock + 1), std::min((int)size, prop.maxThreadsPerBlock) >> > (dev_result, dev_left, dev_right, size);

	hipMemcpy(result, dev_result, size * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(dev_left);
	hipFree(dev_right);
	hipFree(dev_result);
}

__global__ void addMat(double** c, double** a, double** b, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	for (size_t k = 0; k < n; k+= blockDim.x)
	{
		for (size_t l = 0; l < n; l += blockDim.y)
		{
			if (i + k < n && j + l < n)
				c[i + k][j + l] = a[i + k][j + l] + b[i + k][j + l];
		}
	}
}

__global__ void subMat(double** c, double** a, double** b, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	for (size_t k = 0; k < n; k += blockDim.x)
	{
		for (size_t l = 0; l < n; l += blockDim.y)
		{
			if (i + k < n && j + l < n)
				c[i + k][j + l] = a[i + k][j + l] - b[i + k][j + l];
		}
	}
}

__global__ void mulMat(double** c, double** a, double** b, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	for (size_t k = 0; k < n; k += blockDim.x)
	{
		for (size_t l = 0; l < n; l += blockDim.y)
		{
			if (i + k < n && j + l < n)
			{
				c[i + k][j + l] = 0;
				for (size_t m = 0; m < n; m++)
				{
					c[i + k][j + l] += a[i + k][m] * b[m][j + l];
				}
			}
		}
	}
}

__global__ void mulMat(double** c, double** a, double *b, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	for (size_t k = 0; k < n; k += blockDim.x)
	{
		for (size_t l = 0; l < n; l += blockDim.y)
		{
			if (i + k < n && j + l < n)
				c[i + k][j + l] = a[i + k][j + l] * (*b);
		}
	}
}

void CUDA::Matrix::add(double** result, double** left, double** right, size_t size)
{
	double** dev_left = nullptr;
	double** dev_right = nullptr;
	double** dev_result = nullptr;

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	
	hipMalloc((void**)&dev_left, size * sizeof(double*));
	hipMalloc((void**)&dev_right, size * sizeof(double*));
	hipMalloc((void**)&dev_result, size * sizeof(double*));
	
	for (size_t i = 0; i < size; i++)
	{
		hipMalloc((void**)&dev_right[i], size * sizeof(double));
		hipMalloc((void**)&dev_left[i], size * sizeof(double));
		hipMalloc((void**)&dev_result[i], size * sizeof(double));
		
		hipMemcpy(dev_left[i], left[i], size * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(dev_right[i], right[i], size * sizeof(double), hipMemcpyHostToDevice);
	}

	dim3 dimGrid(std::min((int)size, prop.maxGridSize[0]), std::min((int)size, prop.maxGridSize[1]));
	dim3 dimBlock(std::min((int)size, prop.maxThreadsDim[0]), std::min((int)size, prop.maxThreadsDim[1]));

	addMat << < dimGrid, dimBlock >> > (dev_result, dev_left, dev_right, size);
	
	hipMemcpy(result, dev_result, size * sizeof(double*), hipMemcpyDeviceToHost);

	for (size_t i = 0; i < size; i++)
	{
		hipFree(dev_left[i]);
		hipFree(dev_right[i]);
		hipFree(dev_result[i]);
	}
	hipFree(dev_left);
	hipFree(dev_right);
	hipFree(dev_result);
}

void CUDA::Matrix::sub(double** result, double** left, double** right, size_t size)
{
	double** dev_left = nullptr;
	double** dev_right = nullptr;
	double** dev_result = nullptr;

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

	hipMalloc((void**)&dev_left, size * sizeof(double*));
	hipMalloc((void**)&dev_right, size * sizeof(double*));
	hipMalloc((void**)&dev_result, size * sizeof(double*));

	for (size_t i = 0; i < size; i++)
	{
		hipMalloc((void**)&dev_right[i], size * sizeof(double));
		hipMalloc((void**)&dev_left[i], size * sizeof(double));
		hipMalloc((void**)&dev_result[i], size * sizeof(double));

		hipMemcpy(dev_left[i], left[i], size * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(dev_right[i], right[i], size * sizeof(double), hipMemcpyHostToDevice);
	}

	dim3 dimGrid(std::min((int)size, prop.maxGridSize[0]), std::min((int)size, prop.maxGridSize[1]));
	dim3 dimBlock(std::min((int)size, prop.maxThreadsDim[0]), std::min((int)size, prop.maxThreadsDim[1]));

	subMat << < dimGrid, dimBlock >> > (dev_result, dev_left, dev_right, size);

	hipMemcpy(result, dev_result, size * sizeof(double*), hipMemcpyDeviceToHost);

	for (size_t i = 0; i < size; i++)
	{
		hipFree(dev_left[i]);
		hipFree(dev_right[i]);
		hipFree(dev_result[i]);
	}
	hipFree(dev_left);
	hipFree(dev_right);
	hipFree(dev_result);
}

void CUDA::Matrix::mul(double** result, double** left, double** right, size_t size)
{
	double** dev_left = nullptr;
	double** dev_right = nullptr;
	double** dev_result = nullptr;

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

	hipMalloc((void**)&dev_left, size * sizeof(double*));
	hipMalloc((void**)&dev_right, size * sizeof(double*));
	hipMalloc((void**)&dev_result, size * sizeof(double*));

	for (size_t i = 0; i < size; i++)
	{
		hipMalloc((void**)&dev_right[i], size * sizeof(double));
		hipMalloc((void**)&dev_left[i], size * sizeof(double));
		hipMalloc((void**)&dev_result[i], size * sizeof(double));

		hipMemcpy(dev_left[i], left[i], size * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(dev_right[i], right[i], size * sizeof(double), hipMemcpyHostToDevice);
	}

	dim3 dimGrid(std::min((int)size, prop.maxGridSize[0]), std::min((int)size, prop.maxGridSize[1]));
	dim3 dimBlock(std::min((int)size, prop.maxThreadsDim[0]), std::min((int)size, prop.maxThreadsDim[1]));

	mulMat << < dimGrid, dimBlock >> > (dev_result, dev_left, dev_right, size);

	hipMemcpy(result, dev_result, size * sizeof(double*), hipMemcpyDeviceToHost);

	for (size_t i = 0; i < size; i++)
	{
		hipFree(dev_left[i]);
		hipFree(dev_right[i]);
		hipFree(dev_result[i]);
	}
	hipFree(dev_left);
	hipFree(dev_right);
	hipFree(dev_result);
}

void CUDA::Matrix::mul(double** result, double** left, double right, size_t size)
{
	double** dev_left = nullptr;
	double** dev_result = nullptr;
	double* dev_right;

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

	hipMalloc((void**)&dev_left, size * sizeof(double*));
	hipMalloc((void**)&dev_result, size * sizeof(double*));
	hipMalloc((void**)&dev_right, sizeof(double));

	hipMemcpy(dev_right, &right, sizeof(double), hipMemcpyHostToDevice);

	for (size_t i = 0; i < size; i++)
	{
		hipMalloc((void**)&dev_left[i], size * sizeof(double));
		hipMalloc((void**)&dev_result[i], size * sizeof(double));

		hipMemcpy(dev_left[i], left[i], size * sizeof(double), hipMemcpyHostToDevice);
	}

	dim3 dimGrid(std::min((int)size, prop.maxGridSize[0]), std::min((int)size, prop.maxGridSize[1]));
	dim3 dimBlock(std::min((int)size, prop.maxThreadsDim[0]), std::min((int)size, prop.maxThreadsDim[1]));

	mulMat << < dimGrid, dimBlock >> > (dev_result, dev_left, dev_right, size);

	hipMemcpy(result, dev_result, size * sizeof(double*), hipMemcpyDeviceToHost);

	for (size_t i = 0; i < size; i++)
	{
		hipFree(dev_left[i]);
		hipFree(dev_result[i]);
	}
	hipFree(dev_left);
	hipFree(dev_result);
	hipFree(dev_right);
}