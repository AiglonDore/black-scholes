#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "kernel.cuh"

__global__ void addKernel(double* c, const double* a, const double* b, size_t size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
		c[i] = a[i] + b[i];
}

void CUDA::Vector::add(double* result, const double* left, const double* right, size_t size)
{
	double* dev_left = nullptr;
	double* dev_right = nullptr;
	double* dev_result = nullptr;

	hipMalloc((void**)&dev_left, size * sizeof(double));
	hipMalloc((void**)&dev_right, size * sizeof(double));
	hipMalloc((void**)&dev_result, size * sizeof(double));

	hipMemcpy(dev_left, left, size * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_right, right, size * sizeof(double), hipMemcpyHostToDevice);
	
	//addKernel << <(size + 255) / 256, 256 >> > (dev_result, dev_left, dev_right, size);
	
	hipMemcpy(result, dev_result, size * sizeof(double), hipMemcpyDeviceToHost);
	
	hipFree(dev_left);
	hipFree(dev_right);
	hipFree(dev_result);
}